#include "hip/hip_runtime.h"
/* Copyright (c) 2010-2011, Panos Louridas, GRNET S.A.
 
   All rights reserved.
  
   Redistribution and use in source and binary forms, with or without
   modification, are permitted provided that the following conditions
   are met:
 
   * Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.
 
   * Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the
   distribution.
 
   * Neither the name of GRNET S.A, nor the names of its contributors
   may be used to endorse or promote products derived from this
   software without specific prior written permission.
  
   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
   "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
   LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
   FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
   COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
   INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
   (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
   SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
   HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
   STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
   ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
   OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <vector>
#include <map>
#include <math.h>
#include <string>
#include <cstring>
#include <limits>
#include "stdio.h"
#include "table.cuh"


void Table::reset() {
    num_outgoing.clear();
    rows.clear();
    nodes_to_idx.clear();
    idx_to_nodes.clear();
    pr.clear();
}

Table::Table(double a, double c, size_t i, bool t, bool n, string d)
    : trace(t),
      alpha(a),
      convergence(c),
      max_iterations(i),
      delim(d),
      numeric(n) {
}

void Table::reserve(size_t size) {
    num_outgoing.reserve(size);
    rows.reserve(size);
}

const size_t Table::get_num_rows() {
    return rows.size();
}

void Table::set_num_rows(size_t num_rows) {
    num_outgoing.resize(num_rows);
    rows.resize(num_rows);
}

const void Table::error(const char *p,const char *p2) {
    cerr << p <<  ' ' << p2 <<  '\n';
    exit(1);
}

const double Table::get_alpha() {
    return alpha;
}

void Table::set_alpha(double a) {
    alpha = a;
}

const unsigned long Table::get_max_iterations() {
    return max_iterations;
}

void Table::set_max_iterations(unsigned long i) {
    max_iterations = i;
}

const double Table::get_convergence() {
    return convergence;
}

void Table::set_convergence(double c) {
    convergence = c;
}

const vector<double>& Table::get_pagerank() {
    return pr;
}

const string Table::get_node_name(size_t index) {
    if (numeric) {
        stringstream s;
        s << index;
        return s.str();
    } else {
        return idx_to_nodes[index];
    }
}

const map<size_t, string>& Table::get_mapping() {
    return idx_to_nodes;
}

const bool Table::get_trace() {
    return trace;
}

void Table::set_trace(bool t) {
    trace = t;
}

const bool Table::get_numeric() {
    return numeric;
}

void Table::set_numeric(bool n) {
    numeric = n;
}

const string Table::get_delim() {
    return delim;
}

void Table::set_delim(string d) {
    delim = d;
}

/*
 * From a blog post at: http://bit.ly/1QQ3hv
 */
void Table::trim(string &str) {

    size_t startpos = str.find_first_not_of(" \t");

    if (string::npos == startpos) {
        str = "";
    } else {
        str = str.substr(startpos, str.find_last_not_of(" \t") - startpos + 1);
    }
}

size_t Table::insert_mapping(const string &key) {

    size_t index = 0;
    map<string, size_t>::const_iterator i = nodes_to_idx.find(key);
    if (i != nodes_to_idx.end()) {
        index = i->second;
    } else {
        index = nodes_to_idx.size();
        nodes_to_idx.insert(pair<string, size_t>(key, index));
        idx_to_nodes.insert(pair<size_t, string>(index, key));;
    }

    return index;
}

int Table::read_file(const string &filename) {

    pair<map<string, size_t>::iterator, bool> ret;

    reset();
    
    istream *infile;

    if (filename.empty()) {
      infile = &cin;
    } else {
      infile = new ifstream(filename.c_str());
      if (!infile) {
          error("Cannot open file", filename.c_str());
      }
    }
    
    size_t delim_len = delim.length();
    size_t linenum = 0;
    string line; // current line
    while (getline(*infile, line)) {
        string from, to; // from and to fields
        size_t from_idx, to_idx; // indices of from and to nodes
        size_t pos = line.find(delim);
        if (pos != string::npos) {
            from = line.substr(0, pos);
            trim(from);
            if (!numeric) {
                from_idx = insert_mapping(from);
            } else {
                from_idx = strtol(from.c_str(), NULL, 10);
            }
            to = line.substr(pos + delim_len);
            trim(to);
            if (!numeric) {
                to_idx = insert_mapping(to);
            } else {
                to_idx = strtol(to.c_str(), NULL, 10);
            }
            add_arc(from_idx, to_idx);
        }

        linenum++;
        if (linenum && ((linenum % 100000) == 0)) {
            cerr << "read " << linenum << " lines, "
                 << rows.size() << " vertices" << endl;
        }

        from.clear();
        to.clear();
        line.clear();
    }

    cerr << "read " << linenum << " lines, "
         << rows.size() << " vertices" << endl;

    nodes_to_idx.clear();

    if (infile != &cin) {
        delete infile;
    }
    reserve(idx_to_nodes.size());

    //vector<unsigned> row_rep = {0};
    //vector<unsigned> col_rep;
    row_rep.push_back(0);

    for (unsigned nodes = 0; nodes < rows.size(); nodes++){
        acc += rows[nodes].size();
        row_rep.push_back(acc);
        col_rep.insert(col_rep.end(), rows[nodes].begin(), rows[nodes].end());
        //if (rows[nodes].size() == 0)
            //zero_outgoing_idxvec.push_back(nodes);
    }

    return 0;
}

/*
 * Taken from: M. H. Austern, "Why You Shouldn't Use set - and What You Should
 * Use Instead", C++ Report 12:4, April 2000.
 */
template <class Vector, class T>
bool Table::insert_into_vector(Vector& v, const T& t) {
    typename Vector::iterator i = lower_bound(v.begin(), v.end(), t);
    if (i == v.end() || t < *i) {
        v.insert(i, t);
        return true;
    } else {
        return false;
    }
}

bool Table::add_arc(size_t from, size_t to) {

    bool ret = false;
    size_t max_dim = max(from, to);
    if (trace) {
        cout << "checking to add " << from << " => " << to << endl;
    }
    if (rows.size() <= max_dim) {
        max_dim = max_dim + 1;
        if (trace) {
            cout << "resizing rows from " << rows.size() << " to "
                 << max_dim << endl;
        }
        rows.resize(max_dim);
        if (num_outgoing.size() <= max_dim) {
            num_outgoing.resize(max_dim);
        }
    }

    ret = insert_into_vector(rows[to], from);

    if (ret) {
        num_outgoing[from]++;
        if (trace) {
            cout << "added " << from << " => " << to << endl;
        }
    }

    return ret;
}

__global__ void danglingPr();

__global__ void pageRank(
    const size_t* csrRowPtrA, 
    const size_t* csrColPtrA, 
    const size_t* num_outgoing, 
    double* pr, 
    double* old_pr,
    double* diff, 
    const double alpha,
    const double one_Av,
    const double one_Iv,
    const double sum_pr)
    {
    
        unsigned ki = blockIdx.x*blockDim.x + threadIdx.x;
        old_pr[ki] = pr[ki]/sum_pr;
        unsigned row_start = csrRowPtrA[ki];
        unsigned row_end = csrRowPtrA[ki+1];
        //printf ("Thread number %d; Row Start: %d; Row end = %d \n", threadIdx.x, row_start, row_end);
        double h = 0.0;
        for(unsigned i = row_start; i < row_end; i++){
            unsigned ci = csrColPtrA[i];
            double h_v = (num_outgoing[ci])
            ? 1.0 / num_outgoing[ci]
            : 0.0;
            h += h_v * old_pr[ci];
            //printf ("Thread number %d; Column index (pointing at me): %d; h = %f \n", threadIdx.x, ci, h);       
        }
        h *= alpha;
        double this_pr = h + one_Av + one_Iv; 
        pr[ki] = this_pr;
        diff[ki] = fabs(this_pr - old_pr[ki]);
        //pr[ki] = ki;
}

void Table::pagerank() {

    vector<size_t>::iterator ci; // current incoming
    size_t num_rows = rows.size();

    double diff = 1;
    size_t i;
    double sum_pr; // sum of current pagerank vector elements
    double dangling_pr = 0; // sum of current pagerank vector elements for dangling
    			// nodes
    unsigned long num_iterations = 0;
    vector<double> old_pr;



    //print_table();

    //print_outgoing();

    
    if (num_rows == 0) {
        return;
    }
    
    pr.resize(num_rows);

    pr[0] = 1;

    if (trace) {
        print_pagerank();
    }
  
    vector<unsigned> zero_outgoing_idxvec;

    for (size_t k = 0; k < pr.size(); k++) {
        if (num_outgoing[k] == 0) {
            zero_outgoing_idxvec.push_back(k);
        }
    }

    thrust::device_vector<unsigned> d_zero_outgoing_index;
    d_zero_outgoing_index = zero_outgoing_idxvec;

    // allocate num_outgoing, pr, old_pr, csrRowPtrA, csrColPtrA
    size_t int_size = num_rows * sizeof(size_t);
    size_t double_size = num_rows * sizeof(double);
    size_t col_size = col_rep.size() * sizeof(size_t);
    size_t row_size = (row_rep.size() + 1) * sizeof(size_t);
    size_t dangling_size = (zero_outgoing_idxvec.size()) * sizeof(double);

    size_t* d_num_outgoing;
    size_t* d_row_rep;
    size_t* d_col_rep;
    double* d_pr;
    double* d_old_pr;
    double* d_diff;
    double* d_dangling;

    hipMalloc(&d_num_outgoing, int_size);
    hipMalloc(&d_row_rep, row_size);
    hipMalloc(&d_col_rep, col_size);
    hipMalloc(&d_pr, double_size);
    hipMalloc(&d_old_pr, double_size);
    hipMalloc(&d_diff, double_size);
    hipMalloc(&d_dangling, dangling_size);

    hipMemcpy(d_num_outgoing, &num_outgoing[0], int_size, hipMemcpyHostToDevice);
    hipMemcpy(d_row_rep, &row_rep[0], row_size, hipMemcpyHostToDevice);
    hipMemcpy(d_col_rep, &col_rep[0], col_size, hipMemcpyHostToDevice);
    //hipMemcpy(d_old_pr, &pr[0], double_size, hipMemcpyHostToDevice);
    hipMemcpy(d_pr, &pr[0], double_size, hipMemcpyHostToDevice);

    double one_Av;
    double one_Iv = (1 - alpha) / num_rows;

    while (diff > convergence && num_iterations < max_iterations) {
        
        thrust::gather(thrust::device,
            d_zero_outgoing_index.begin(), d_zero_outgoing_index.end(),
            d_pr,
            d_dangling
        );

        dangling_pr = thrust::reduce(thrust::device, d_dangling, d_dangling + zero_outgoing_idxvec.size());
        sum_pr = thrust::reduce(thrust::device, d_pr, d_pr + num_rows); 
                             
        one_Av = alpha * dangling_pr / num_rows;

        pageRank<<<38000, 500>>>(
            d_row_rep,
            d_col_rep, 
            d_num_outgoing,
            d_pr,
            d_old_pr,
            d_diff,
            alpha,
            one_Av,
            one_Iv,
            sum_pr);

        hipDeviceSynchronize();

        diff = thrust::reduce(thrust::device, d_diff, d_diff + num_rows);

        num_iterations++;

    }

    hipMemcpy(&pr[0], d_pr, double_size, hipMemcpyDeviceToHost);

    hipFree(d_col_rep);
    hipFree(d_row_rep);
    hipFree(d_num_outgoing);
    hipFree(d_old_pr);
    hipFree(d_pr);
    hipFree(d_dangling);
}

const void Table::print_params(ostream& out) {
    out << "alpha = " << alpha << " convergence = " << convergence
        << " max_iterations = " << max_iterations
        << " numeric = " << numeric
        << " delimiter = '" << delim << "'" << endl;
}

const void Table::print_table() {
    vector< vector<size_t> >::iterator cr;
    vector<size_t>::iterator cc; // current column

    size_t i = 0;
    for (cr = rows.begin(); cr != rows.end(); cr++) {
        cout << i << ":[ ";
        for (cc = cr->begin(); cc != cr->end(); cc++) {
            if (numeric) {
                cout << *cc << " ";
            } else {
                cout << idx_to_nodes[*cc] << " ";
            }
        }
        cout << "]" << endl;
        i++;
    }
}

const void Table::print_outgoing() {
    vector<size_t>::iterator cn;

    cout << "[ ";
    for (cn = num_outgoing.begin(); cn != num_outgoing.end(); cn++) {
        cout << *cn << " ";
    }
    cout << "]" << endl;

}

const void Table::print_row_col() {
    vector<size_t>::iterator cn;

    cout << "Row: \n [ ";
    for (cn = row_rep.begin(); cn != row_rep.end(); cn++) {
        cout << *cn << " ";
    }
    cout << "]" << endl;

    cout << "Column: \n [ ";
    for (cn = col_rep.begin(); cn != col_rep.end(); cn++) {
        cout << *cn << " ";
    }
    cout << "]" << endl;

}

const void Table::print_pagerank() {

    vector<double>::iterator cr;
    double sum = 0;

    cout.precision(numeric_limits<double>::digits10);
    
    cout << "(" << pr.size() << ") " << "[ ";
    for (cr = pr.begin(); cr != pr.end(); cr++) {
        cout << *cr << " ";
        sum += *cr;
        cout << "s = " << sum << " ";
    }
    cout << "] "<< sum << endl;
}

const void Table::print_pagerank_v() {

    size_t i;
    size_t num_rows = pr.size();
    double sum = 0;
    
    cout.precision(numeric_limits<double>::digits10);

    for (i = 0; i < num_rows; i++) {
        if (!numeric) {
            cout << idx_to_nodes[i] << " = " << pr[i] << endl;
        } else {
            cout << i << " = " << pr[i] << endl;
        }
        sum += pr[i];
    }
    cerr << "s = " << sum << " " << endl;
}
